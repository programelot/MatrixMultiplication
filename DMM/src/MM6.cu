#include "hip/hip_runtime.h"
#include "GPUdebug.hpp"
#include "common.hpp"

using namespace std;

__global__ void MM_Kernel(dataType* matrix, dataType* input, dataType* res,
        size_t sizeX, size_t sizeRange, size_t sizeY, size_t resUnit, size_t matUnit, size_t inputUnit){
    size_t li = threadIdx.x; size_t bi = blockIdx.x; size_t bis = blockDim.x;
    size_t lj = threadIdx.y; size_t bj = blockIdx.y; size_t bjs = blockDim.y;
    size_t globalRowIdx = li + bi * bis;
    size_t globalColIdx = lj + bj * bjs;
    dataType acc = 0;
    __shared__ dataType subMatrix[subBlockSize][subBlockSize];
    __shared__ dataType subInput[subBlockSize][subBlockSize];
    for(int k = 0; k < (sizeRange + subBlockSize - 1)/subBlockSize; ++k){
        size_t localMatRowIdx = globalRowIdx;
        size_t localMatColIdx = lj + subBlockSize * k;
        size_t localInputRowIdx = li + subBlockSize * k;
        size_t localInputColIdx = globalColIdx;
        if(localMatRowIdx < sizeX && localMatColIdx < sizeRange) {
            subMatrix[li][lj] = matrix[localMatRowIdx * matUnit + localMatColIdx];
        }
        if(localInputRowIdx < sizeRange && localInputColIdx < sizeY){
            subInput[lj][li]  = input [localInputRowIdx * inputUnit + localInputColIdx];
        }
        __syncthreads();
        for(int lk = 0; lk < subBlockSize; ++lk){
            if(subBlockSize * k + lk < sizeRange){
                acc += subMatrix[li][lk] * subInput[lj][lk];
            }
        }
        __syncthreads();
    }
    if(globalRowIdx < sizeX && globalColIdx < sizeY)
        res[globalRowIdx * resUnit + globalColIdx] = acc;
}

float gemm(dataType* matrix, dataType* input, dataType* res,
    const size_t rowSize, const size_t rangeSize, const size_t colSize){
    const size_t rowNumMat = rowSize;
    const size_t rowNumInp = rangeSize;
    const size_t rowNumRes = rowSize;
    const size_t colNumMat = rangeSize;
    const size_t colNumInp = colSize;
    const size_t colNumRes = colSize;
    dataType *matrix_GPU;
    dataType *input_GPU;
    dataType *res_GPU;
    HANDLE_ERROR(hipMalloc(&matrix_GPU, sizeof(dataType)*rowNumMat*colNumMat));
    HANDLE_ERROR(hipMalloc(&input_GPU, sizeof(dataType)*rowNumInp*colNumInp));
    HANDLE_ERROR(hipMalloc(&res_GPU, sizeof(dataType)*rowNumRes*colNumRes));
    HANDLE_ERROR(hipMemcpy(matrix_GPU, matrix, sizeof(dataType) * colNumMat * rowNumMat, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(input_GPU, input, sizeof(dataType) * colNumInp * rowNumInp, hipMemcpyHostToDevice));

    dim3 threadDim(rowNumRes > subBlockSize ? subBlockSize : rowNumRes , colNumRes > subBlockSize ? subBlockSize : colNumRes);
    dim3 blockDim((rowNumRes + subBlockSize - 1)/subBlockSize , (colNumRes + subBlockSize - 1)/subBlockSize);
    hipEvent_t start, stop;
    float milliseconds = 0;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));
    MM_Kernel<<<blockDim, threadDim>>>(matrix_GPU, input_GPU, res_GPU, rowSize, rangeSize, colSize, colSize, rangeSize, colSize);
    HANDLE_ERROR(hipMemcpy(res, res_GPU, sizeof(dataType) * colNumRes * rowNumRes, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    HANDLE_ERROR(hipFree(matrix_GPU));
    HANDLE_ERROR(hipFree(input_GPU));
    HANDLE_ERROR(hipFree(res_GPU));
    return milliseconds;
}
