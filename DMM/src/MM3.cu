#include "hip/hip_runtime.h"
#include "GPUdebug.hpp"
#include "common.hpp"

using namespace std;

__global__ void MM_Kernel(dataType* matrix, dataType* input, dataType* res,
        size_t sizeX, size_t sizeRange, size_t sizeY, size_t resUnit, size_t matUnit, size_t inputUnit){
    size_t global = threadIdx.x + blockIdx.x * blockDim.x;
    size_t i = global/sizeY;
    size_t j = global%sizeY;
    global = i * resUnit + j;
    if(global > sizeX * sizeY) return;
    dataType acc = 0;
    for(int k = 0; k < sizeRange; ++k){
        acc +=  matrix[i * matUnit + k] * input[k * inputUnit + j];
    }
    res[global] = acc;
}

float gemm(dataType* matrix, dataType* input, dataType* res,
    const size_t rowSize, const size_t rangeSize, const size_t colSize){
    const size_t rowNumMat = rowSize;
    const size_t rowNumInp = rangeSize;
    const size_t rowNumRes = rowSize;
    const size_t colNumMat = rangeSize;
    const size_t colNumInp = colSize;
    const size_t colNumRes = colSize;
    dataType *matrix_GPU;
    dataType *input_GPU;
    dataType *res_GPU;
    HANDLE_ERROR(hipMalloc(&matrix_GPU, sizeof(dataType)*rowNumMat*colNumMat));
    HANDLE_ERROR(hipMalloc(&input_GPU,  sizeof(int)*rowNumInp*colNumInp));
    HANDLE_ERROR(hipMalloc(&res_GPU,    sizeof(int)*rowNumRes*colNumRes));
    HANDLE_ERROR(hipMemcpy(matrix_GPU, matrix, sizeof(dataType) * colNumMat * rowNumMat, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(input_GPU, input, sizeof(dataType) * colNumInp * rowNumInp, hipMemcpyHostToDevice));

    int work = rowNumRes * colNumRes;
    hipEvent_t start, stop;
    float milliseconds = 0;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));
    MM_Kernel<<<(work + MAXTHREADSIZE - 1)/MAXTHREADSIZE, work > MAXTHREADSIZE? MAXTHREADSIZE : work >>>(matrix_GPU, input_GPU, res_GPU, rowSize, rangeSize, colSize, colSize, rangeSize, colSize);
    HANDLE_ERROR(hipMemcpy(res, res_GPU, sizeof(dataType) * colNumRes * rowNumRes, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    HANDLE_ERROR(hipFree(matrix_GPU));
    HANDLE_ERROR(hipFree(input_GPU));
    HANDLE_ERROR(hipFree(res_GPU));
    return milliseconds;
}
