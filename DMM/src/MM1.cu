#include "GPUdebug.hpp"
#include "common.hpp"

using namespace std;

float gemm(dataType* matrix, dataType* input, dataType* res,
    const size_t rowSize, const size_t rangeSize, const size_t colSize){
    const size_t rowNumMat = rowSize;
    const size_t rowNumInp = rangeSize;
    const size_t rowNumRes = rowSize;
    const size_t colNumMat = rangeSize;
    const size_t colNumInp = colSize;
    const size_t colNumRes = colSize;
    hipEvent_t start, stop;
    float milliseconds = 0;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));
    for(int i = 0; i < rowNumRes; ++i){
        for(int j = 0; j < colNumRes; ++j){
            res[i * rowSize + j] = 0;
            for(int k = 0; k < colNumMat; ++k){
                res[i * rowSize + j] += matrix[i * rowSize + k] * input[k * rangeSize + j];
            }
        }
    }
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    return milliseconds;
}
