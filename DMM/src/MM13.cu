#include "GPUdebug.hpp"
#include "common.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;

float gemm(dataType* matrix, dataType* input, dataType* res,
    const size_t rowSize, const size_t rangeSize, const size_t colSize){
    const size_t rowNumMat = rowSize;
    const size_t rowNumInp = rangeSize;
    const size_t rowNumRes = rowSize;
    const size_t colNumMat = rangeSize;
    const size_t colNumInp = colSize;
    const size_t colNumRes = colSize;
    dataType *matrix_GPU;
    dataType *input_GPU;
    dataType *res_GPU;
    HANDLE_ERROR(hipMalloc(&matrix_GPU, sizeof(dataType) * rowNumMat * colNumMat));
    HANDLE_ERROR(hipMalloc(&input_GPU, sizeof(dataType)  * rowNumInp * colNumInp));
    HANDLE_ERROR(hipMalloc(&res_GPU, sizeof(dataType)    * rowNumRes * colNumRes));
    HANDLE_ERROR(hipMemcpy(matrix_GPU, matrix, sizeof(dataType) * rowNumMat * colNumMat, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(input_GPU,  input,  sizeof(dataType) * rowNumInp * colNumInp, hipMemcpyHostToDevice));


    hipEvent_t start, stop;
    float milliseconds = 0;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));
    HANDLE_ERROR(hipDeviceSynchronize());
    float alpha = 1, beta = 0;

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle));
    
    HANDLE_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    colSize, rowSize, rangeSize, &alpha, input_GPU, colNumInp, matrix_GPU, colNumMat, &beta, res_GPU, colNumRes)); 
    HANDLE_ERROR(hipMemcpy(res, res_GPU, sizeof(dataType) * colNumRes * rowNumRes, hipMemcpyDeviceToHost));
    HANDLE_CUBLAS_ERROR(hipblasDestroy(handle));

    HANDLE_ERROR(hipDeviceSynchronize());
    
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    HANDLE_ERROR(hipFree(matrix_GPU));
    HANDLE_ERROR(hipFree(input_GPU));
    HANDLE_ERROR(hipFree(res_GPU));
    return milliseconds;
}
