#include "hip/hip_runtime.h"
#include "GPUdebug.hpp"
#include "common.hpp"

using namespace std;

__global__ void MM_Kernel(dataType* matrix, dataType* input, dataType* res,
        size_t sizeX, size_t sizeRange, size_t sizeY, size_t resUnit, size_t matUnit, size_t inputUnit){
    size_t global = threadIdx.x + blockIdx.x * blockDim.x;
    size_t i = global/sizeY;
    size_t j = global%sizeY;
    global = i * resUnit + j;
    if(i >= sizeX || j >= sizeY) return;
    dataType acc = 0;
    for(int k = 0; k < sizeRange; ++k){
        acc +=  matrix[i * matUnit + k] * input[k * inputUnit + j];
    }
    res[global] = acc;
}

__global__ void SetPlusM_Kernel(dataType *from, size_t fromUnit,
            dataType *to,   size_t toUnit,
            size_t height, size_t width){
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= height || j >= width) return;
    to[i * toUnit + j] = from[i * fromUnit + j];
}

__global__ void SetMinusM_Kernel(dataType *from, size_t fromUnit,
            dataType *to,   size_t toUnit,
            size_t height, size_t width){
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= height || j >= width) return;
    to[i * toUnit + j] = -from[i * fromUnit + j];
}

__global__ void SetZero_Kernel(dataType *mat, size_t Unit, size_t length){
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= length) return;
    mat[i * Unit] = 0;
}

__global__ void AddM_Kernel(dataType *from, size_t fromUnit,
            dataType *to,   size_t toUnit,
            size_t height, size_t width){
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= height || j >= width) return;
    to[i * toUnit + j] += from[i * fromUnit + j];
}

__global__ void SubM_Kernel(dataType *from, size_t fromUnit,
            dataType *to,   size_t toUnit,
            size_t height, size_t width){
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    size_t j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= height || j >= width) return;
    to[i * toUnit + j] -= from[i * fromUnit + j];
}

bool isBaseCase(size_t sizeX, size_t sizeRange, size_t sizeY){
    return sizeX * sizeRange<= 1048576 || sizeRange * sizeY <= 1048576 || sizeX * sizeY <= 1048576 ||
        sizeX <= 32 || sizeRange <= 32 || sizeY <= 32;
    //return sizeX <= 1024 || sizeRange <= 1024 || sizeY <= 1024;
}

//Copy array from "from" to "to"
//Index will starts from "fromOffset" and "toOffset"
//It will copy width in a row for height times with each corresponding units.
void addMat(dataType *from, size_t fromUnit,
            dataType *to, size_t toUnit,
            size_t height, size_t width, bool plus){
    dim3 threadDim(height > subBlockSize ? subBlockSize : height , width > subBlockSize ? subBlockSize : width);
    dim3 blockDim((height + subBlockSize - 1)/subBlockSize , (width + subBlockSize - 1)/subBlockSize);
    if(plus){
        AddM_Kernel<<<blockDim, threadDim>>>(from, fromUnit, to, toUnit, height, width);
    }
    else{
        SubM_Kernel<<<blockDim, threadDim>>>(from, fromUnit, to, toUnit, height, width);
    }
}

//Set matrix value <from> to <to>
//Fill zero for one line of row or col or both if it needed.
void setMat(dataType *from, size_t fromUnit,
            dataType *to, size_t toUnit,
            size_t height, size_t width, 
            bool extendHeight, bool extendWidth, bool plus){
    dim3 threadDim(height > subBlockSize ? subBlockSize : height , width > subBlockSize ? subBlockSize : width);
    dim3 blockDim((height + subBlockSize - 1)/subBlockSize , (width + subBlockSize - 1)/subBlockSize);

    
    if(plus){
        SetPlusM_Kernel<<<blockDim, threadDim>>>(from, fromUnit, to, toUnit, height, width);
    }
    else{
        SetMinusM_Kernel<<<blockDim, threadDim>>>(from, fromUnit, to, toUnit, height, width);
    }

    if(extendHeight){
        // SetZero_Kernel<<<((width + (extendWidth ? 1 : 0)) + MAXTHREADSIZE - 1) / MAXTHREADSIZE,
        //                   (width + (extendWidth ? 1 : 0)) > MAXTHREADSIZE? MAXTHREADSIZE : (width + (extendWidth ? 1 : 0))>>>
        //                   (to + (height * toUnit), 1, (width + (extendWidth ? 1 : 0)));
        HANDLE_ERROR(hipMemset(to + (height * toUnit), 0,  sizeof(dataType) * (width + (extendWidth ? 1 : 0))));
    }

    if(extendWidth){
        SetZero_Kernel<<<(height + MAXTHREADSIZE - 1) / MAXTHREADSIZE, height > MAXTHREADSIZE? MAXTHREADSIZE : height>>>(to + width, toUnit, height);
    }
}

void gemm(  dataType * matrix, dataType * input, dataType * result,
            size_t matUnit, size_t inpUnit, size_t resUnit,
            dataType * auxRes, size_t auxResUnit,
            size_t sizeX, size_t sizeRange, size_t sizeY ){

    bool baseCase = isBaseCase(sizeX, sizeRange, sizeY);
    if(baseCase){
        size_t work = sizeX * sizeY;
        MM_Kernel<<<(work + MAXTHREADSIZE - 1)/MAXTHREADSIZE, work > MAXTHREADSIZE? MAXTHREADSIZE : work >>>(matrix, input, result, sizeX, sizeRange, sizeY, resUnit, matUnit, inpUnit);
    }
    else{
        const size_t rowNumMat = sizeX;
        const size_t colNumMat = sizeRange;
        const size_t rowNumInp = sizeRange;
        const size_t colNumInp = sizeY;
        const size_t rowNumRes = sizeX;
        const size_t colNumRes = sizeY;
        
        const size_t rowNumAuxmat = (rowNumMat + 1)/2;
        const size_t colNumAuxmat = (colNumMat + 1)/2;
        const size_t rowNumAuxinp = (rowNumInp + 1)/2;
        const size_t colNumAuxinp = (colNumInp + 1)/2;
        const size_t rowNumAuxres = (rowNumRes + 1)/2;
        const size_t colNumAuxres = (colNumRes + 1)/2;

        const size_t rowNumAuxmatLeft = rowNumMat - rowNumAuxmat;
        const size_t colNumAuxmatLeft = colNumMat - colNumAuxmat;
        const size_t rowNumAuxinpLeft = rowNumInp - rowNumAuxinp;
        const size_t colNumAuxinpLeft = colNumInp - colNumAuxinp;
        const size_t rowNumAuxresLeft = rowNumRes - rowNumAuxres;
        const size_t colNumAuxresLeft = colNumRes - colNumAuxres;
        
        const size_t auxMatSize = rowNumAuxmat * colNumAuxmat;
        const size_t auxInpSize = rowNumAuxinp * colNumAuxinp;
        const size_t auxResSize = rowNumAuxres * colNumAuxres;

        const size_t auxSizeX     = rowNumAuxres;
        const size_t auxSizeRange = colNumAuxmat;
        const size_t auxSizeY     = colNumAuxres;
        
        const size_t auxSizeXLeft     = rowNumAuxresLeft;
        const size_t auxSizeRangeLeft = colNumAuxmatLeft;
        const size_t auxSizeYLeft     = colNumAuxresLeft;
        
        const size_t rowNumAuxauxmat = (rowNumAuxmat + 1)/2;
        const size_t colNumAuxauxmat = (colNumAuxmat + 1)/2;
        const size_t rowNumAuxauxinp = (rowNumAuxinp + 1)/2;
        const size_t colNumAuxauxinp = (colNumAuxinp + 1)/2;
        const size_t rowNumAuxauxres = (rowNumAuxres + 1)/2;
        const size_t colNumAuxauxres = (colNumAuxres + 1)/2;

        dataType* auxAuxRes = auxRes + auxResSize;

        dataType* mat_11 = matrix;
        dataType* mat_12 = mat_11 + colNumAuxmat;
        dataType* mat_21 = mat_11 + matUnit * rowNumAuxmat;
        dataType* mat_22 = mat_21 + colNumAuxmat;

        dataType* inp_11 = input;
        dataType* inp_12 = inp_11 + colNumAuxinp;
        dataType* inp_21 = inp_11 + inpUnit * rowNumAuxinp;
        dataType* inp_22 = inp_21 + colNumAuxinp;

        dataType* res_11 = result;
        dataType* res_12 = res_11 + colNumAuxres;
        dataType* res_21 = res_11 + resUnit * rowNumAuxres;
        dataType* res_22 = res_21 + colNumAuxres;

        //////////////////////////////////////////////////
        // mat_11 inp_11
        // Positive : res_11
        //////////////////////////////////////////////////

        gemm( mat_11, inp_11, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeX, auxSizeRange, auxSizeY);

        setMat( auxRes, auxResUnit, 
                res_11, resUnit,
                rowNumAuxres, colNumAuxres, 
                false, false, true);
        //////////////////////////////////////////////////
        // mat_12 inp_21
        // Positive : res_11
        //////////////////////////////////////////////////

        gemm( mat_12, inp_21, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeX, auxSizeRangeLeft, auxSizeY);
              
        addMat( auxRes, auxResUnit, 
                res_11, resUnit,
                rowNumAuxres, colNumAuxres, true);

        //////////////////////////////////////////////////
        // mat_11 inp_12
        // Positive : res_12
        //////////////////////////////////////////////////

        gemm( mat_11, inp_12, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeX, auxSizeRange, auxSizeYLeft);

        setMat( auxRes, auxResUnit, 
                res_12, resUnit,
                rowNumAuxres, colNumAuxresLeft,
                false, false, true);

        //////////////////////////////////////////////////
        // mat_12 inp_22
        // Positive : res_12
        //////////////////////////////////////////////////

        gemm( mat_12, inp_22, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeX, auxSizeRangeLeft, auxSizeYLeft);
              
        addMat( auxRes, auxResUnit, 
                res_12, resUnit,
                rowNumAuxres, colNumAuxresLeft, true);

        //////////////////////////////////////////////////
        // mat_21 inp_11
        // Positive : res_21
        //////////////////////////////////////////////////

        gemm( mat_21, inp_11, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeXLeft, auxSizeRange, auxSizeY);

        setMat( auxRes, auxResUnit, 
                res_21, resUnit,
                rowNumAuxresLeft, colNumAuxres,
                false, false, true);

        //////////////////////////////////////////////////
        // mat_22 inp_21
        // Positive : res_21
        //////////////////////////////////////////////////

        gemm( mat_22, inp_21, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeXLeft, auxSizeRangeLeft, auxSizeY);
              
        addMat( auxRes, auxResUnit, 
                res_21, resUnit,
                rowNumAuxresLeft, colNumAuxres, true);

        //////////////////////////////////////////////////
        // mat_21 inp_12
        // Positive : res_22
        //////////////////////////////////////////////////

        gemm( mat_21, inp_12, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeXLeft, auxSizeRange, auxSizeYLeft);

        setMat( auxRes, auxResUnit, 
                res_22, resUnit,
                rowNumAuxresLeft, colNumAuxresLeft,
                false, false, true);

        //////////////////////////////////////////////////
        // mat_22 inp_22
        // Positive : res_22
        //////////////////////////////////////////////////

        gemm( mat_22, inp_22, auxRes,
              matUnit, inpUnit, auxResUnit,
              auxAuxRes, colNumAuxauxres,
              auxSizeXLeft, auxSizeRangeLeft, auxSizeYLeft);
              
        addMat( auxRes, auxResUnit, 
                res_22, resUnit,
                rowNumAuxresLeft, colNumAuxresLeft, true);
    }
}


float gemm(dataType* matrix, dataType* input, dataType* res,
    const size_t rowSize, const size_t rangeSize, const size_t colSize){
    dataType *matrix_GPU, *input_GPU, *res_GPU;
    
    const size_t rowNumMat = rowSize;
    const size_t colNumMat = rangeSize;
    const size_t rowNumInp = rangeSize;
    const size_t colNumInp = colSize;
    const size_t rowNumRes = rowSize;
    const size_t colNumRes = colSize;

    //Get auxiliary requirement size
    size_t auxSize_Res = 0;
    {
        size_t auxRow = rowSize;
        size_t auxRange = rangeSize;
        size_t auxCol = colSize;
        while(!isBaseCase(auxRow, auxRange, auxCol)){//Minimum matrix multiplication is 1024
            auxRow = (auxRow + 1)/2;
            auxCol = (auxCol + 1)/2;
            auxRange = (auxRange + 1)/2;
            
            auxSize_Res += auxRow * auxCol;
        }
    }

    HANDLE_ERROR(hipMalloc(&matrix_GPU, sizeof(dataType)*(rowNumMat*colNumMat)));
    HANDLE_ERROR(hipMalloc(&input_GPU,  sizeof(dataType)*(rowNumInp*colNumInp)));
    HANDLE_ERROR(hipMalloc(&res_GPU,    sizeof(dataType)*(rowNumRes*colNumRes + auxSize_Res)));
    HANDLE_ERROR(hipMemcpy(matrix_GPU, matrix, sizeof(dataType) * colNumMat * rowNumMat, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(input_GPU, input, sizeof(dataType) * colNumInp * rowNumInp, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    float milliseconds = 0;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));

    const size_t resSize = colNumRes * rowNumRes;

    gemm(   matrix_GPU, input_GPU, res_GPU,
            colNumMat, colNumInp, colNumRes,
            res_GPU + resSize, (colNumRes + 1)/2,
            rowNumMat, colNumMat, colNumInp);
    HANDLE_ERROR(hipMemcpy(res, res_GPU, sizeof(dataType) * colNumRes * rowNumRes, hipMemcpyDeviceToHost));
    
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    HANDLE_ERROR(hipFree(matrix_GPU));
    HANDLE_ERROR(hipFree(input_GPU));
    HANDLE_ERROR(hipFree(res_GPU));
    return milliseconds;
}